#include "hip/hip_runtime.h"
#include "MCsrc_cuda.hh"
#include <ios>
#include <iostream>
#include <istream>
#include <limits>
#include <map>
#include <vector>

#include "TClonesArray.h"
#include "TString.h"
#include "TFile.h"
#include "TTree.h"
#include "TTreeReader.h"
#include "TTreePlayer.h"
#include "TTreeReaderValue.h"
#include "TSystem.h"
#include "TH1F.h"
#include "TCanvas.h"

#include "ATEvent.hh"
#include "ATPad.hh"
#include "ATHit.hh"
#include "ATHoughSpace.hh"
#include "ATHoughSpaceLine.hh"
#include "ATHoughSpaceCircle.hh"

#include "FairRootManager.h"
#include "FairLogger.h"
#include "FairRun.h"
#include "FairRunAna.h"


__global__ void test_GPU_function(int* a, int* b, Int_t N){
  int th = blockIdx.x*blockDim.x+threadIdx.x;
  if(th < N){
    b[th] = 0;
    for(int i=0;i<N;i++)
      b[th] += a[i]; 
  }
}


Int_t main()
{

    gSystem->Load("libATTPCReco.so");
    
    FairRunAna* run = new FairRunAna(); //Forcing a dummy run

    TString workdir = getenv("VMCWORKDIR");
  
    /***********************************************/
    /* Test GPU code                               */
    Int_t N = 100000;
    size_t size = N*sizeof(int);

    int* h_a = (int*)malloc(size);
    for(int i=0; i<N; i++){
      h_a[i] = 1;
    }
    int* h_b = (int*)malloc(size);
   
    int* d_a;
    hipMalloc(&d_a,size);
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    int* d_b;
    hipMalloc(&d_b,size);

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock -1) / threadsPerBlock;
    test_GPU_function<<<numBlocks,threadsPerBlock>>>(d_a,d_b,N);

    hipMemcpy(h_b,d_b,size,hipMemcpyDeviceToHost);
  
    //std::vector<int> h_b_vec = h_b;
    for(int i=0;i<N;i++){
      std::cout << "h_b[" << i << "] = " << h_b[i]  << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    /**********************************************/


    /*TString FileNameHead = "output";
    TString FilePath = workdir + "/macro/Unpack_GETDecoder2/";
    TString FileNameTail = ".root";
    TString FileName     = FilePath + FileNameHead + FileNameTail;

    std::cout<<" Opening File : "<<FileName.Data()<<std::endl;
    TFile* file = new TFile(FileName.Data(),"READ");

    TTree* tree = (TTree*) file -> Get("cbmsim");
    Int_t nEvents = tree -> GetEntries();
    std::cout<<" Number of events : "<<nEvents<<std::endl;

    TTreeReader Reader1("cbmsim", file);
    TTreeReaderValue<TClonesArray> eventArray(Reader1, "ATEventH");
    TTreeReaderValue<TClonesArray> houghArray(Reader1, "ATHough");

          while (Reader1.Next()) {


              ATEvent* event = (ATEvent*) eventArray->At(0);
              Int_t nHits = event->GetNumHits();
              std::vector<ATHit>* hitArray = event->GetHitArray(); //Not working!
              event->GetHitArrayObj();
              //std::cout<<event->GetHitPadMult(0)<<std::endl;
              //std::cout<<event->GetEventID()<<std::endl;
              hitArray->size();

              std::vector<ATHit*>* hitbuff = new std::vector<ATHit*>; // Working!

              //std::vector<ATEvent*> test;
              //test.push_back(event);

                    for(Int_t iHit=0; iHit<nHits; iHit++){
                      ATHit hit = event->GetHit(iHit);
                      TVector3 hitPos = hit.GetPosition();
                      hitbuff->push_back(&hit);


                    }

              //std::cout<<hitbuff->size()<<std::endl;

              ATHoughSpaceCircle* fHoughSpaceCircle  = dynamic_cast<ATHoughSpaceCircle*> (houghArray->At(0));
              //if(!fHoughSpaceCircle) std::cout<<" Warning : Failed casting "<<std::endl;
              std::cout<<fHoughSpaceCircle->GetYCenter()<<std::endl;


          }

  //#pragma omp parallel for ordered schedule(dynamic,1)
  //for(Int_t i=0;i<100;i++)std::cout<<" Hello ATTPCer! "<<std::endl;

  */
   return 0;

}
